#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "playground/matmul.hpp"
#include "playground/static.hpp"
#include "playground/system.hpp"

namespace playground
{
PLAYGROUND_MATMUL_SIG(float32_t, 11, M, N, K, A, B, C)
{
    const float32_t Alpha = 1.0f;
    const float32_t Beta = 0.0f;
    // hipblasSgemm(s_getCublasHandle<float32_t>(), HIPBLAS_OP_N, HIPBLAS_OP_N, N, M,
    //             K, &Alpha, B, N, A, K, &Beta, C, N);

    hipblasGemmEx(s_getCublasHandle<float32_t>(), HIPBLAS_OP_N, HIPBLAS_OP_N, N, M,
                 K, &Alpha, B, HIP_R_32F, N, A, HIP_R_32F, K, &Beta, C,
                 HIP_R_32F, N, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}
}  // namespace playground