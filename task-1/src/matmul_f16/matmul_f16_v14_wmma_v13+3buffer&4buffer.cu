#include "hip/hip_runtime.h"
#include "playground/matmul.hpp"
#include "playground/system.hpp"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <mma.h>

using namespace nvcuda;

#define OFFSET(row, col, ld) ((row) * (ld) + (col))

#define ASYNC_COPY_TO_SHARED(dst, src, size)                                   \
    asm volatile("cp.async.ca.shared.global [%0], [%1], " #size ";\n"          \
                 :                                                             \
                 : "r"(dst), "l"(src))

#define COMMIT_ASYNC_GROUP() asm volatile("cp.async.commit_group;\n" ::)

#define WAIT_ASYNC_GROUP() asm volatile("cp.async.wait_group 0;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n)                                                \
    asm volatile("cp.async.wait_group %0;\n" ::"n"(n))

#define HOST_DEVICE_INLINE __device__ __host__ inline
HOST_DEVICE_INLINE
int div_ceil(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

namespace playground
{
__global__ void hgemm_v13_triple_buffered(const float16_t* __restrict__ a,
                                          const float16_t* __restrict__ b,
                                          float16_t* __restrict__ c,
                                          const int M, const int N, const int K)
{
    const int BM = 128;
    const int BN = 256;
    const int BK = 32;

    int by = blockIdx.y;
    int bx = blockIdx.z * gridDim.x + blockIdx.x;
    if (bx >= N / BN || by >= M / BM) {
        return;
    }
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    // const int tid = threadIdx.x + threadIdx.y * blockDim.x +
    //                 threadIdx.z * blockDim.x * blockDim.y;
    int wid = tid >> 5;

    const int APAD = 8;
    const int BPAD = 8;

    extern __shared__ float16_t smem[];
    float16_t* s_a = smem;
    float16_t* s_b = smem + 3 * BM * (BK + APAD);
    int s_a_db_offset = BM * (BK + APAD);
    int s_b_db_offset = BK * (BN + BPAD);

    wmma::fragment<wmma::matrix_a, 16, 16, 16, float16_t, wmma::row_major>
        frag_a[2][4];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, float16_t, wmma::row_major>
        frag_b[2][4];
    wmma::fragment<wmma::accumulator, 16, 16, 16, float16_t> frag_c[4][4];

#pragma unroll
    for (int i = 0; i < 4; i++) {
#pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::fill_fragment(frag_c[i][j], 0.0);
        }
    }

    int load_a_smem_m = (tid >> 2) << 1;
    int load_a_smem_k = (tid & 3) << 3;
    int load_b_smem_k = (tid >> 5) << 2;
    int load_b_smem_n = (tid & 31) << 3;

    int s_a_base_addr = __cvta_generic_to_shared(s_a);
    int s_b_base_addr = __cvta_generic_to_shared(s_b);

    int load_a_smem_addr_0 =
        s_a_base_addr +
        OFFSET(load_a_smem_m, load_a_smem_k, BK + APAD) * sizeof(float16_t);
    int load_a_smem_addr_1 =
        load_a_smem_addr_0 + (BK + APAD) * sizeof(float16_t);
    int load_b_smem_addr_0 =
        s_b_base_addr +
        OFFSET(load_b_smem_k, load_b_smem_n, BN + BPAD) * sizeof(float16_t);
    int load_b_smem_addr_1 =
        load_b_smem_addr_0 + (BN + BPAD) * sizeof(float16_t);
    int load_b_smem_addr_2 =
        load_b_smem_addr_0 + 2 * (BN + BPAD) * sizeof(float16_t);
    int load_b_smem_addr_3 =
        load_b_smem_addr_0 + 3 * (BN + BPAD) * sizeof(float16_t);

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_smem_k, K);
    int load_b_gmem_addr = OFFSET(load_b_smem_k, load_b_gmem_n, N);

    int comp_c_frag_m = wid & 1;
    int comp_c_frag_n = wid >> 1;

    {
        // buffer 0
        ASYNC_COPY_TO_SHARED(load_a_smem_addr_0, &a[load_a_gmem_addr], 16);
        ASYNC_COPY_TO_SHARED(load_a_smem_addr_1, &a[load_a_gmem_addr + K], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_0, &b[load_b_gmem_addr], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_1, &b[load_b_gmem_addr + N], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_2, &b[load_b_gmem_addr + 2 * N],
                             16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_3, &b[load_b_gmem_addr + 3 * N],
                             16);


        load_a_gmem_addr += BK;
        load_b_gmem_addr += BK * N;

        // buffer1
        ASYNC_COPY_TO_SHARED(load_a_smem_addr_0 +
                                 s_a_db_offset * (int) sizeof(float16_t),
                             &a[load_a_gmem_addr], 16);
        ASYNC_COPY_TO_SHARED(load_a_smem_addr_1 +
                                 s_a_db_offset * (int) sizeof(float16_t),
                             &a[load_a_gmem_addr + K], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_0 +
                                 s_b_db_offset * (int) sizeof(float16_t),
                             &b[load_b_gmem_addr], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_1 +
                                 s_b_db_offset * (int) sizeof(float16_t),
                             &b[load_b_gmem_addr + N], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_2 +
                                 s_b_db_offset * (int) sizeof(float16_t),
                             &b[load_b_gmem_addr + 2 * N], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_3 +
                                 s_b_db_offset * (int) sizeof(float16_t),
                             &b[load_b_gmem_addr + 3 * N], 16);

        COMMIT_ASYNC_GROUP();
        CP_ASYNC_WAIT_GROUP(1);

        __syncthreads();
    }

#pragma unroll 32
    for (int bk = 2; bk < div_ceil(K, BK); bk++) {

        // int smem_sel = (bk & 1) ^ 1;
        // int smem_sel_next = ((bk - 1) & 1) ^ 1;
        int smem_sel_next = (bk % 3);
        int smem_sel = ((bk + 1) % 3);


        ASYNC_COPY_TO_SHARED(load_a_smem_addr_0 + smem_sel_next *
                                                      s_a_db_offset *
                                                      (int) sizeof(float16_t),
                             &a[load_a_gmem_addr], 16);
        ASYNC_COPY_TO_SHARED(load_a_smem_addr_1 + smem_sel_next *
                                                      s_a_db_offset *
                                                      (int) sizeof(float16_t),
                             &a[load_a_gmem_addr + K], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_0 + smem_sel_next *
                                                      s_b_db_offset *
                                                      (int) sizeof(float16_t),
                             &b[load_b_gmem_addr], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_1 + smem_sel_next *
                                                      s_b_db_offset *
                                                      (int) sizeof(float16_t),
                             &b[load_b_gmem_addr + N], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_2 + smem_sel_next *
                                                      s_b_db_offset *
                                                      (int) sizeof(float16_t),
                             &b[load_b_gmem_addr + 2 * N], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_3 + smem_sel_next *
                                                      s_b_db_offset *
                                                      (int) sizeof(float16_t),
                             &b[load_b_gmem_addr + 3 * N], 16);
        COMMIT_ASYNC_GROUP();
        load_a_gmem_addr += BK;
        load_b_gmem_addr += BK * N;
        wmma::load_matrix_sync(frag_a[0][0],
                               &s_a[smem_sel * s_a_db_offset +
                                    (comp_c_frag_m * 64) * (BK + APAD) + 0],
                               BK + APAD);
        wmma::load_matrix_sync(
            frag_a[0][1],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 16) * (BK + APAD) + 0],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[0][2],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 32) * (BK + APAD) + 0],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[0][3],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 48) * (BK + APAD) + 0],
            BK + APAD);
        wmma::load_matrix_sync(frag_a[1][0],
                               &s_a[smem_sel * s_a_db_offset +
                                    (comp_c_frag_m * 64) * (BK + APAD) + 16],
                               BK + APAD);
        wmma::load_matrix_sync(
            frag_a[1][1],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 16) * (BK + APAD) + 16],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[1][2],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 32) * (BK + APAD) + 16],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[1][3],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 48) * (BK + APAD) + 16],
            BK + APAD);

        wmma::load_matrix_sync(
            frag_b[0][0], &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64],
            BN + BPAD);
        wmma::load_matrix_sync(
            frag_b[0][1],
            &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64 + 16],
            BN + BPAD);
        wmma::load_matrix_sync(
            frag_b[0][2],
            &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64 + 32],
            BN + BPAD);
        wmma::load_matrix_sync(
            frag_b[0][3],
            &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64 + 48],
            BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][0],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64],
                               BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][1],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64 + 16],
                               BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][2],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64 + 32],
                               BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][3],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64 + 48],
                               BN + BPAD);

#pragma unroll
        for (int i = 0; i < 4; i++) {
#pragma unroll
            for (int j = 0; j < 4; j++) {
                wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j],
                               frag_c[i][j]);
                wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j],
                               frag_c[i][j]);
            }
        }

        CP_ASYNC_WAIT_GROUP(1);
        __syncthreads();
        
    }
    CP_ASYNC_WAIT_GROUP(0);
    __syncthreads();
    // int smem_sel = ((K / BK) & 1) ^ 1;
#pragma unroll
    for (int k = 0; k < 2; k++) {
        const int smem_sel = ((div_ceil(K, BK) - 2 + k) % 3);
        wmma::load_matrix_sync(frag_a[0][0],
                               &s_a[smem_sel * s_a_db_offset +
                                    (comp_c_frag_m * 64) * (BK + APAD) + 0],
                               BK + APAD);
        wmma::load_matrix_sync(
            frag_a[0][1],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 16) * (BK + APAD) + 0],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[0][2],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 32) * (BK + APAD) + 0],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[0][3],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 48) * (BK + APAD) + 0],
            BK + APAD);
        wmma::load_matrix_sync(frag_a[1][0],
                               &s_a[smem_sel * s_a_db_offset +
                                    (comp_c_frag_m * 64) * (BK + APAD) + 16],
                               BK + APAD);
        wmma::load_matrix_sync(
            frag_a[1][1],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 16) * (BK + APAD) + 16],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[1][2],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 32) * (BK + APAD) + 16],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[1][3],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 48) * (BK + APAD) + 16],
            BK + APAD);

        wmma::load_matrix_sync(
            frag_b[0][0], &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64],
            BN + BPAD);
        wmma::load_matrix_sync(
            frag_b[0][1],
            &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64 + 16],
            BN + BPAD);
        wmma::load_matrix_sync(
            frag_b[0][2],
            &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64 + 32],
            BN + BPAD);
        wmma::load_matrix_sync(
            frag_b[0][3],
            &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64 + 48],
            BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][0],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64],
                               BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][1],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64 + 16],
                               BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][2],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64 + 32],
                               BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][3],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64 + 48],
                               BN + BPAD);

#pragma unroll
        for (int i = 0; i < 4; i++) {
#pragma unroll
            for (int j = 0; j < 4; j++) {
                wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j],
                               frag_c[i][j]);
                wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j],
                               frag_c[i][j]);
            }
        }
        // __syncthreads();
    }

    

    int store_c_gmem_m = by * BM + comp_c_frag_m * 64;
    int store_c_gmem_n = bx * BN + comp_c_frag_n * 64;
    int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
#pragma unroll
    for (int i = 0; i < 4; i++) {
#pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::store_matrix_sync(&c[store_c_gmem_addr + i * 16 * N + j * 16],
                                    frag_c[i][j], N, wmma::mem_row_major);
        }
    }
}

__global__ void hgemm_v13_quad_buffered(const float16_t* __restrict__ a,
                                          const float16_t* __restrict__ b,
                                          float16_t* __restrict__ c,
                                          const int M, const int N, const int K)
{
    const int BM = 128;
    const int BN = 256;
    const int BK = 32;
    const int NUM_K_TILES = K / BK;
    int by = blockIdx.y;
    int bx = blockIdx.z * gridDim.x + blockIdx.x;
    if (bx >= N / BN || by >= M / BM) {
        return;
    }
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    // const int tid = threadIdx.x + threadIdx.y * blockDim.x +
    //                 threadIdx.z * blockDim.x * blockDim.y;
    int wid = tid >> 5;

    const int APAD = 8;
    const int BPAD = 8;

    extern __shared__ float16_t smem[];
    float16_t* s_a = smem;
    float16_t* s_b = smem + 4 * BM * (BK + APAD);
    int s_a_db_offset = BM * (BK + APAD);
    int s_b_db_offset = BK * (BN + BPAD);

    wmma::fragment<wmma::matrix_a, 16, 16, 16, float16_t, wmma::row_major>
        frag_a[2][4];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, float16_t, wmma::row_major>
        frag_b[2][4];
    wmma::fragment<wmma::accumulator, 16, 16, 16, float16_t> frag_c[4][4];

#pragma unroll
    for (int i = 0; i < 4; i++) {
#pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::fill_fragment(frag_c[i][j], 0.0);
        }
    }

    int load_a_smem_m = (tid >> 2) << 1;
    int load_a_smem_k = (tid & 3) << 3;
    int load_b_smem_k = (tid >> 5) << 2;
    int load_b_smem_n = (tid & 31) << 3;

    int s_a_base_addr = __cvta_generic_to_shared(s_a);
    int s_b_base_addr = __cvta_generic_to_shared(s_b);

    int load_a_smem_addr_0 =
        s_a_base_addr +
        OFFSET(load_a_smem_m, load_a_smem_k, BK + APAD) * sizeof(float16_t);
    int load_a_smem_addr_1 =
        load_a_smem_addr_0 + (BK + APAD) * sizeof(float16_t);
    int load_b_smem_addr_0 =
        s_b_base_addr +
        OFFSET(load_b_smem_k, load_b_smem_n, BN + BPAD) * sizeof(float16_t);
    int load_b_smem_addr_1 =
        load_b_smem_addr_0 + (BN + BPAD) * sizeof(float16_t);
    int load_b_smem_addr_2 =
        load_b_smem_addr_0 + 2 * (BN + BPAD) * sizeof(float16_t);
    int load_b_smem_addr_3 =
        load_b_smem_addr_0 + 3 * (BN + BPAD) * sizeof(float16_t);

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_smem_k, K);
    int load_b_gmem_addr = OFFSET(load_b_smem_k, load_b_gmem_n, N);

    int comp_c_frag_m = wid & 1;
    int comp_c_frag_n = wid >> 1;

    // Preload 3 buffers
#pragma unroll
    for (int buf = 0; buf < 3; buf++) {
        ASYNC_COPY_TO_SHARED(load_a_smem_addr_0 +
                                 buf * s_a_db_offset * (int) sizeof(float16_t),
                             &a[load_a_gmem_addr], 16);
        ASYNC_COPY_TO_SHARED(load_a_smem_addr_1 +
                                 buf * s_a_db_offset * (int) sizeof(float16_t),
                             &a[load_a_gmem_addr + K], 16);

        ASYNC_COPY_TO_SHARED(load_b_smem_addr_0 +
                                 buf * s_b_db_offset * (int) sizeof(float16_t),
                             &b[load_b_gmem_addr], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_1 +
                                 buf * s_b_db_offset * (int) sizeof(float16_t),
                             &b[load_b_gmem_addr + N], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_2 +
                                 buf * s_b_db_offset * (int) sizeof(float16_t),
                             &b[load_b_gmem_addr + 2 * N], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_3 +
                                 buf * s_b_db_offset * (int) sizeof(float16_t),
                             &b[load_b_gmem_addr + 3 * N], 16);

        COMMIT_ASYNC_GROUP();

        load_a_gmem_addr += BK;
        load_b_gmem_addr += BK * N;
    }
    CP_ASYNC_WAIT_GROUP(2);
    __syncthreads();

#pragma unroll 24
    for (int bk = 3; bk < NUM_K_TILES; bk++) {

        // int smem_sel = (bk & 1) ^ 1;
        // int smem_sel_next = ((bk - 1) & 1) ^ 1;
        int smem_sel_next = bk & 3;
        int smem_sel = (bk + 1) & 3;

        ASYNC_COPY_TO_SHARED(load_a_smem_addr_0 + smem_sel_next *
                                                      s_a_db_offset *
                                                      (int) sizeof(float16_t),
                             &a[load_a_gmem_addr], 16);
        ASYNC_COPY_TO_SHARED(load_a_smem_addr_1 + smem_sel_next *
                                                      s_a_db_offset *
                                                      (int) sizeof(float16_t),
                             &a[load_a_gmem_addr + K], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_0 + smem_sel_next *
                                                      s_b_db_offset *
                                                      (int) sizeof(float16_t),
                             &b[load_b_gmem_addr], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_1 + smem_sel_next *
                                                      s_b_db_offset *
                                                      (int) sizeof(float16_t),
                             &b[load_b_gmem_addr + N], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_2 + smem_sel_next *
                                                      s_b_db_offset *
                                                      (int) sizeof(float16_t),
                             &b[load_b_gmem_addr + 2 * N], 16);
        ASYNC_COPY_TO_SHARED(load_b_smem_addr_3 + smem_sel_next *
                                                      s_b_db_offset *
                                                      (int) sizeof(float16_t),
                             &b[load_b_gmem_addr + 3 * N], 16);
        load_a_gmem_addr += BK;
        load_b_gmem_addr += BK * N;
        wmma::load_matrix_sync(frag_a[0][0],
                               &s_a[smem_sel * s_a_db_offset +
                                    (comp_c_frag_m * 64) * (BK + APAD) + 0],
                               BK + APAD);
        wmma::load_matrix_sync(
            frag_a[0][1],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 16) * (BK + APAD) + 0],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[0][2],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 32) * (BK + APAD) + 0],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[0][3],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 48) * (BK + APAD) + 0],
            BK + APAD);
        wmma::load_matrix_sync(frag_a[1][0],
                               &s_a[smem_sel * s_a_db_offset +
                                    (comp_c_frag_m * 64) * (BK + APAD) + 16],
                               BK + APAD);
        wmma::load_matrix_sync(
            frag_a[1][1],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 16) * (BK + APAD) + 16],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[1][2],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 32) * (BK + APAD) + 16],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[1][3],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 48) * (BK + APAD) + 16],
            BK + APAD);

        wmma::load_matrix_sync(
            frag_b[0][0], &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64],
            BN + BPAD);
        wmma::load_matrix_sync(
            frag_b[0][1],
            &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64 + 16],
            BN + BPAD);
        wmma::load_matrix_sync(
            frag_b[0][2],
            &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64 + 32],
            BN + BPAD);
        wmma::load_matrix_sync(
            frag_b[0][3],
            &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64 + 48],
            BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][0],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64],
                               BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][1],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64 + 16],
                               BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][2],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64 + 32],
                               BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][3],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64 + 48],
                               BN + BPAD);

#pragma unroll
        for (int i = 0; i < 4; i++) {
#pragma unroll
            for (int j = 0; j < 4; j++) {
                wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j],
                               frag_c[i][j]);
                wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j],
                               frag_c[i][j]);
            }
        }
        COMMIT_ASYNC_GROUP();
        CP_ASYNC_WAIT_GROUP(2);

        __syncthreads();
    }

    CP_ASYNC_WAIT_GROUP(0);
    // __syncthreads();
    // int smem_sel = ((K / BK) & 1) ^ 1;
#pragma unroll
    for (int k = 0; k < 3; k++) {
        const int smem_sel = ((NUM_K_TILES - 3 + k) & 3);
        wmma::load_matrix_sync(frag_a[0][0],
                               &s_a[smem_sel * s_a_db_offset +
                                    (comp_c_frag_m * 64) * (BK + APAD) + 0],
                               BK + APAD);
        wmma::load_matrix_sync(
            frag_a[0][1],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 16) * (BK + APAD) + 0],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[0][2],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 32) * (BK + APAD) + 0],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[0][3],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 48) * (BK + APAD) + 0],
            BK + APAD);
        wmma::load_matrix_sync(frag_a[1][0],
                               &s_a[smem_sel * s_a_db_offset +
                                    (comp_c_frag_m * 64) * (BK + APAD) + 16],
                               BK + APAD);
        wmma::load_matrix_sync(
            frag_a[1][1],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 16) * (BK + APAD) + 16],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[1][2],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 32) * (BK + APAD) + 16],
            BK + APAD);
        wmma::load_matrix_sync(
            frag_a[1][3],
            &s_a[smem_sel * s_a_db_offset +
                 (comp_c_frag_m * 64 + 48) * (BK + APAD) + 16],
            BK + APAD);

        wmma::load_matrix_sync(
            frag_b[0][0], &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64],
            BN + BPAD);
        wmma::load_matrix_sync(
            frag_b[0][1],
            &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64 + 16],
            BN + BPAD);
        wmma::load_matrix_sync(
            frag_b[0][2],
            &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64 + 32],
            BN + BPAD);
        wmma::load_matrix_sync(
            frag_b[0][3],
            &s_b[smem_sel * s_b_db_offset + comp_c_frag_n * 64 + 48],
            BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][0],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64],
                               BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][1],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64 + 16],
                               BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][2],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64 + 32],
                               BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][3],
                               &s_b[smem_sel * s_b_db_offset +
                                    16 * (BN + BPAD) + comp_c_frag_n * 64 + 48],
                               BN + BPAD);

#pragma unroll
        for (int i = 0; i < 4; i++) {
#pragma unroll
            for (int j = 0; j < 4; j++) {
                wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j],
                               frag_c[i][j]);
                wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j],
                               frag_c[i][j]);
            }
        }
        __syncthreads();
    }

    int store_c_gmem_m = by * BM + comp_c_frag_m * 64;
    int store_c_gmem_n = bx * BN + comp_c_frag_n * 64;
    int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
#pragma unroll
    for (int i = 0; i < 4; i++) {
#pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::store_matrix_sync(&c[store_c_gmem_addr + i * 16 * N + j * 16],
                                    frag_c[i][j], N, wmma::mem_row_major);
        }
    }
}

PLAYGROUND_MATMUL_DEC(float16_t, 14, M, N, K, A, B, C)
{
    const int BM = 128, BN = 256, BK = 32;
    // const int K_STAGE = 4;
    dim3 blockDim(32, 8);
    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;

    const int NSPLIT = 4096;
    int split_num = (N + NSPLIT - 1) / NSPLIT;
    dim3 gridDim((BX + split_num - 1) / split_num, BY, split_num);

    hipFuncSetAttribute(reinterpret_cast<const void*>(hgemm_v13_quad_buffered),
                         hipFuncAttributeMaxDynamicSharedMemorySize, 131072);

    unsigned int dsmem =
        4 * (BM * (BK + 8) + BK * (BN + 8)) * sizeof(float16_t);
    hgemm_v13_quad_buffered<<<gridDim, blockDim, dsmem>>>(A, B, C, M, N, K);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }
}
}  // namespace playground