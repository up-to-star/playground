#include "hip/hip_runtime.h"
#include "playground/system.hpp"
#include "playground/matmul.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mma.h>

using namespace nvcuda;

#define HOST_DEVICE_INLINE __device__ __host__ inline
#define WARP_SIZE  32
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n)                                                 \
    asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
#define CP_ASYNC_CA(dst, src, bytes)                                           \
    asm volatile(                                                              \
        "cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst),     \
        "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes)                                           \
    asm volatile(                                                              \
        "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst),     \
        "l"(src), "n"(bytes))

HOST_DEVICE_INLINE
int div_ceil(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

namespace playground
{

__device__ __forceinline__ int2 swizzle_block_zorder(int bx, int by,
                                                     int gridDimX)
{
    unsigned int morton = 0;
    for (int i = 0; i < (sizeof(unsigned int) * 8 / 2); i++) {
        morton |= (bx & (1 << i)) << i | (by & (1 << i)) << (i + 1);
    }
    int swizzled_bx = morton % gridDimX;
    int swizzled_by = morton / gridDimX;
    return make_int2(swizzled_bx, swizzled_by);
}

__device__ __forceinline__ int2 swizzle_block_tiled(int bx, int by,
                                                    int gridDimX, int gridDimY,
                                                    int tile_size = 4)
{
    int tile_x = bx / tile_size;
    int tile_y = by / tile_size;
    int inner_x = bx % tile_size;
    int inner_y = by % tile_size;
    int new_bx =
        tile_y * tile_size + inner_x;  // 交换 tile_x 和 tile_y 以改变访问顺序
    int new_by = tile_x * tile_size + inner_y;
    return make_int2(new_bx % gridDimX, new_by % gridDimY);
}

template <typename DType, const int WMMA_M = 16, const int WMMA_N = 16,
          const int WMMA_K = 16, const int WMMA_TILE_M = 4,
          const int WMMA_TILE_N = 2, const int WARP_TILE_M = 2,
          const int WARP_TILE_N = 4, const int A_PAD = 0, const int B_PAD = 0,
          const int K_STAGES = 2, const bool BLOCK_SWIZZLE = false>
__global__ void __launch_bounds__(256) hgemm_wmma16x16x16_mma4x2_warp2x4(const DType* __restrict__ A,
                                                  const DType* __restrict__ B,
                                                  DType* __restrict__ C,
                                                  const int M, const int N,
                                                  const int K)
{

    // int2 swizzled_block =
    //     swizzle_block_tiled(blockIdx.x, blockIdx.y, gridDim.x, gridDim.y);
    // const int bx = swizzled_block.x;
    // const int by = swizzled_block.y;

    // int2 swizzled_block =
    //     swizzle_block_zorder(blockIdx.x, blockIdx.y, gridDim.x);
    // const int bx = swizzled_block.x;
    // const int by = swizzled_block.y;
    const int bx = blockIdx.x + ((int) BLOCK_SWIZZLE) * blockIdx.z *
    gridDim.x, by = blockIdx.y;
    // const int bx = blockIdx.x, by = blockIdx.y;
    const int NUM_K_TILES = div_ceil(K, WMMA_K);
    constexpr int BM = WMMA_M * WMMA_TILE_M * WARP_TILE_M;
    constexpr int BN = WMMA_N * WMMA_TILE_N * WARP_TILE_N;
    constexpr int BK = WMMA_K;

    __shared__ DType sa[K_STAGES][BM][BK + A_PAD], sb[K_STAGES][BK][BN + B_PAD];

    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int warp_id = tid / WARP_SIZE;
    const int warp_m = warp_id >> 1;
    const int warp_n = warp_id & 1;

    const int load_a_smem_m = tid >> 1;
    const int load_a_smem_k = (tid & 1) << 3;
    const int load_b_smem_k = tid >> 4;
    const int load_b_smem_n = (tid & 15) << 3;

    const int load_a_gmem_m = by * BM + load_a_smem_m;
    const int load_b_gmem_n = bx * BN + load_b_smem_n;

    if (load_a_gmem_m >= M || load_b_gmem_n >= N) {
        return;
    }

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, DType>
        c_frag[WARP_TILE_M][WARP_TILE_N];
#pragma unroll
    for (int i = 0; i < WARP_TILE_M; i++) {
#pragma unroll
        for (int j = 0; j < WARP_TILE_N; j++) {
            wmma::fill_fragment(c_frag[i][j], 0.0);
        }
    }

    constexpr int s_a_stage_offset = BM * (A_PAD + BK);
    constexpr int s_b_stage_offset = BK * (B_PAD + BN);
    uint32_t smem_a_base_ptr = __cvta_generic_to_shared(sa);
    uint32_t smem_b_base_ptr = __cvta_generic_to_shared(sb);

    // 预加载前 k - 1 个数据
    {
#pragma unroll
        for (int k = 0; k < (K_STAGES - 1); k++) {
            const int load_a_gmem_k = k * WMMA_K + load_a_smem_k;
            const int load_a_gmem_addr = load_a_gmem_m * K + load_a_gmem_k;
            const int load_b_gmem_k = k * WMMA_K + load_b_smem_k;
            const int load_b_gmem_addr = load_b_gmem_k * N + load_b_gmem_n;

            uint32_t load_a_smem_ptr =
                smem_a_base_ptr +
                 (k * s_a_stage_offset + load_a_smem_m * (BK + A_PAD) +
                  load_a_smem_k) *
                     sizeof(DType);
            CP_ASYNC_CG(load_a_smem_ptr, &A[load_a_gmem_addr], 16);

            uint32_t load_b_smem_ptr =
                smem_b_base_ptr +
                (k * s_b_stage_offset + load_b_smem_k * (BN + B_PAD) +
                 load_b_smem_n) *
                    sizeof(DType);
            CP_ASYNC_CG(load_b_smem_ptr, &B[load_b_gmem_addr], 16);
            CP_ASYNC_COMMIT_GROUP();
        }
        CP_ASYNC_WAIT_GROUP(K_STAGES - 2);
        __syncthreads();
    }

#pragma unroll
    for (int k = K_STAGES - 1; k < NUM_K_TILES; k++) {
        const int smem_sel = (k + 1) % K_STAGES;
        const int smem_sel_next = k % K_STAGES;

        const int load_a_gmem_k = k * WMMA_K + load_a_smem_k;
        const int load_a_gmem_addr = load_a_gmem_m * K + load_a_gmem_k;
        const int load_b_gmem_k = k * WMMA_K + load_b_smem_k;
        const int load_b_gmem_addr = load_b_gmem_k * N + load_b_gmem_n;

        uint32_t load_a_smem_ptr =
            smem_a_base_ptr + (smem_sel_next * s_a_stage_offset +
                               load_a_smem_m * (BK + A_PAD) + load_a_smem_k) *
                                  sizeof(DType);
        CP_ASYNC_CG(load_a_smem_ptr, &A[load_a_gmem_addr], 16);

        uint32_t load_b_smem_ptr =
            smem_b_base_ptr + (smem_sel_next * s_b_stage_offset +
                               load_b_smem_k * (BN + B_PAD) + load_b_smem_n) *
                                  sizeof(DType);
        CP_ASYNC_CG(load_b_smem_ptr, &B[load_b_gmem_addr], 16);
        CP_ASYNC_COMMIT_GROUP();

        wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, DType,
                       wmma::row_major>
            a_frag[WARP_TILE_M];
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, DType,
                       wmma::row_major>
            b_frag[WARP_TILE_N];

#pragma unroll
        for (int i = 0; i < WARP_TILE_M; i++) {
            const int warp_a_smem_m =
                warp_m * WARP_TILE_M * WMMA_M + i * WMMA_M;
            wmma::load_matrix_sync(a_frag[i], &sa[smem_sel][warp_a_smem_m][0], BK + A_PAD);
        }

#pragma unroll
        for (int j = 0; j < WARP_TILE_N; j++) {
            const int warp_b_smem_n =
                warp_n * WARP_TILE_N * WMMA_N + j * WMMA_N;
            wmma::load_matrix_sync(b_frag[j], &sb[smem_sel][0][warp_b_smem_n],
                                   BN + B_PAD);
        }

#pragma unroll
        for (int i = 0; i < WARP_TILE_M; i++) {
#pragma unroll
            for (int j = 0; j < WARP_TILE_N; j++) {
                wmma::mma_sync(c_frag[i][j], a_frag[i], b_frag[j], c_frag[i][j]);
            }
        }
        CP_ASYNC_WAIT_GROUP(K_STAGES - 2);
        __syncthreads();
    }

    if (K_STAGES - 2 > 0) {
        CP_ASYNC_WAIT_GROUP(0);
        __syncthreads();
    }

    // 处理最后 (K_STAGE-1) k iters.
    {
#pragma unroll
        for (int k = 0; k < K_STAGES - 1; k++) {
            const int stage_sel = ((NUM_K_TILES - (K_STAGES - 1) + k) % K_STAGES);
            wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, DType,
                           wmma::row_major>
                a_frag[WARP_TILE_M];
            wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, DType,
                           wmma::row_major>
                b_frag[WARP_TILE_N];

#pragma unroll
            for (int i = 0; i < WARP_TILE_M; i++) {
                const int warp_a_smem_m =
                    warp_m * WARP_TILE_M * WMMA_M + i * WMMA_M;
                wmma::load_matrix_sync(
                    a_frag[i], &sa[stage_sel][warp_a_smem_m][0], BK + A_PAD);
            }

#pragma unroll
            for (int j = 0; j < WARP_TILE_N; j++) {
                const int warp_b_smem_n =
                    warp_n * WARP_TILE_N * WMMA_N + j * WMMA_N;
                wmma::load_matrix_sync(
                    b_frag[j], &sb[stage_sel][0][warp_b_smem_n], BN + B_PAD);
            }

#pragma unroll
            for (int i = 0; i < WARP_TILE_M; i++) {
#pragma unroll
                for (int j = 0; j < WARP_TILE_N; j++) {
                    wmma::mma_sync(c_frag[i][j], a_frag[i], b_frag[j], c_frag[i][j]);
                }
            }
        }
    }

#pragma unroll
    for (int i = 0; i < WARP_TILE_M; i++) {
#pragma unroll
        for (int j = 0; j < WARP_TILE_N; j++) {
            const int store_c_gmem_m =
                by * BM + warp_m * WARP_TILE_M * WMMA_M + i * WMMA_M;
            const int store_c_gmem_n =
                bx * BN + warp_n * WARP_TILE_N * WMMA_N + j * WMMA_N;
            wmma::store_matrix_sync(C + store_c_gmem_m * N + store_c_gmem_n,
                                    c_frag[i][j], N, wmma::mem_row_major);
        }
    }
}



PLAYGROUND_MATMUL_DEC(float16_t, 11, M, N, K, A, B, C)
{
    const int BM = 128, BN = 128;
    const int WMMA_M = 16, WMMA_N = 16, WMMA_K = 16;
    const int WMMA_TILE_M = 4, WMMA_TILE_N = 2;
    const int WARP_TILE_M = 2, WARP_TILE_N = 4;
    const int A_PAD = 8, B_PAD = 8;
    dim3 blockDim(256);
    const int BX = div_ceil(N, BN), BY = div_ceil(M, BM);
    const int NSPLIT = 2048;
    int split_num = (N + NSPLIT - 1) / NSPLIT;
    dim3 gridDim((BX + split_num - 1) / split_num, BY, split_num);
    // dim3 gridDim(BX, BY);
    hgemm_wmma16x16x16_mma4x2_warp2x4<float16_t, WMMA_M, WMMA_N, WMMA_K,
                                      WMMA_TILE_M, WMMA_TILE_N, WARP_TILE_M,
                                      WARP_TILE_N, A_PAD, B_PAD, 3, true>
        <<<gridDim, blockDim>>>(A, B, C, M, N, K);
}

}