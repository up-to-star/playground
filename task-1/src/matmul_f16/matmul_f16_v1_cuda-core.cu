#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/library_types.h>

#include "playground/matmul.hpp"
#include "playground/static.hpp"
#include "playground/system.hpp"

namespace playground
{
PLAYGROUND_MATMUL_SIG(float16_t, 11, M, N, K, A, B, C)
{
    const float16_t Alpha = 1.0f;
    const float16_t Beta = 0.0f;
    hipblasGemmEx(s_getCublasHandle<float16_t>(), HIPBLAS_OP_N, HIPBLAS_OP_N, N, M,
                 K, &Alpha, B, HIP_R_16F, N, A, HIP_R_16F, K, &Beta, C,
                 HIP_R_16F, N, HIP_R_16F, HIPBLAS_GEMM_DEFAULT);
}
}  // namespace playground
