#include "hip/hip_runtime.h"
#include "playground/common.hpp"
#include "playground/matmul.hpp"
#include "playground/system.hpp"


#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mma.h>


namespace playground
{

#define BLOCK_ROWS 256
#define BLOCK_COLS 128

#define WARP_ROWS 64
#define WARP_COLS 64

#define BLOCK_ROW_WARPS 2  // BLOCK_COLS / WARP_COLS
#define BLOCK_COL_WARPS 4  // BLOCK_ROWS / WARP_ROWS

#define BLOCK_ROW_TILES 16  // BLOCK_COLS / MMA_N
#define BLOCK_COL_TILES 16  // BLOCK_ROWS / MMA_M

#define WARP_ROW_TILES 8  // WARP_COLS / MMA_N
#define WARP_COL_TILES 4  // WARP_ROWS / MMA_M

#define WARP_SIZE 32
#define WARPS_PER_BLOCK 8      // BLOCK_ROW_WARPS * BLOCK_COL_WARPS
#define THREADS_PER_BLOCK 256  // WARP_SIZE * WARPS_PER_BLOCK

#define CHUNK_K 2  // 32 / MMA_K

#define THREAD_COPY_BYTES 16

#define CHUNK_LINE_BYTES 64  // CHUNK_K * MMA_K * sizeof(float16_t)
#define CHUNK_COPY_LINES_PER_WARP                                             \
    8  // WARP_SIZE * THREAD_COPY_BYTES / CHUNK_LINE_BYTES
#define CHUNK_COPY_LINE_LANES 4  // WARP_SIZE / CHUNK_COPY_LINES_PER_WARP

#define AB_SMEM_STRIDE 32  // CHUNK_K * MMA_K

#define C_SMEM_STRIDE 128  // BLOCK_COLS
#define C_SMEM_OFFSET 64   // WARP_COLS

#define BLOCK_STRIDE 16

#define SMEM_BANK_ROWS 2  // 32 * 4 / (AB_SMEM_STRIDE * sizeof(float16_t))

#define PERMUTED_OFFSET 8
#define PERMUTED_COLS 4

#define K_STAGE 4

template <const int MMA_M = 16, const int MMA_N = 8, const int MMA_K = 16>
__global__ void hgemm_mma_stage_v2(const float16_t* __restrict__ A,
                                   const float16_t* __restrict__ B,
                                   float16_t* __restrict__ C, const int M,
                                   const int N, const int K)
{
    const size_t M_tiles = div_ceil(M, MMA_M);
    const size_t N_tiles = div_ceil(N, MMA_N);
    const size_t K_tiles = div_ceil(K, MMA_K);

    const size_t block_tile_i =
        (blockIdx.z % 2) ? ((gridDim.y - blockIdx.y - 1) * BLOCK_COL_TILES)
                         : (blockIdx.y * BLOCK_COL_TILES);
    const size_t block_tile_j =
        (blockIdx.z * gridDim.x + blockIdx.x) * BLOCK_ROW_TILES;

    if (block_tile_i >= M_tiles || block_tile_j >= N_tiles) {
        return;
    }

    extern __shared__ float16_t smem[][AB_SMEM_STRIDE];

    const size_t tid = threadIdx.y * blockDim.x + threadIdx.x;
    const size_t warp_id = tid / WARP_SIZE;
    const size_t lane_id = tid % WARP_SIZE;

    constexpr size_t B_smem_idx_off = BLOCK_ROWS;
    // 单个流水线阶段的总行数
    constexpr size_t smem_stage_off = BLOCK_ROWS + BLOCK_COLS;

    float16_t* smem_warp_tile_row_ptr =
        &smem[0][0] + (warp_id / BLOCK_ROW_WARPS) * C_SMEM_STRIDE * WARP_ROWS;
    // 每个warp处理2个MMA tile
    const float16_t* smem_warp_stream_ptr =
        &smem[0][0] + warp_id * MMA_M * 2 * C_SMEM_STRIDE;

    const size_t gmem_idx =
        (block_tile_i + warp_id * 2) * MMA_M * N + block_tile_j * MMA_N;
    const float16_t* src_gmem_warp_stream_ptr = &C[gmem_idx];

    uint32_t RC[WARP_COL_TILES][WARP_ROW_TILES][2];

#pragma unroll
    for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            RC[i][j][0] = 0;
            RC[i][j][1] = 0;
        }
    }

    const float16_t* A_warp_ptr = &A[block_tile_i * MMA_M * K] +
                             BLOCK_ROWS / WARPS_PER_BLOCK * K * warp_id;
    const float16_t* B_warp_ptr = &B[block_tile_j * MMA_N * K] +
                             BLOCK_COLS / WARPS_PER_BLOCK * K * warp_id;

    constexpr size_t A_smem_iters =
        BLOCK_ROWS / (CHUNK_COPY_LINES_PER_WARP * WARPS_PER_BLOCK);
    constexpr size_t B_smem_iters =
        BLOCK_COLS / (CHUNK_COPY_LINES_PER_WARP * WARPS_PER_BLOCK);

    size_t smem_store_idx = 0;
    size_t smem_load_idx = 0;

    size_t smem_store_off = 0;
    size_t smem_load_off = 0;

    size_t A_smem_idx = 0;
    int4* A_lane_ptr = nullptr;

    size_t B_smem_idx = 0;
    int4* B_lane_ptr = nullptr;

    A_smem_idx = smem_store_off + BLOCK_ROWS / WARPS_PER_BLOCK * warp_id;
    A_lane_ptr = (int4*) (A_warp_ptr + (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                 (lane_id % CHUNK_COPY_LINE_LANES);
    A_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
    for (size_t i = 0; i < A_smem_iters; ++i) {
        uint32_t A_smem_lane_addr =
            __cvta_generic_to_shared(&smem[A_smem_idx][0]) +
            ((lane_id % CHUNK_COPY_LINE_LANES +
              (A_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) /
                  SMEM_BANK_ROWS) %
             CHUNK_COPY_LINE_LANES) *
                THREAD_COPY_BYTES;

        CP_ASYNC_CG(A_smem_lane_addr, A_lane_ptr, THREAD_COPY_BYTES);

        A_lane_ptr =
            (int4*) ((float16_t*) A_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
        A_smem_idx += CHUNK_COPY_LINES_PER_WARP;
    }

    B_smem_idx = smem_store_off + B_smem_idx_off +
                 BLOCK_COLS / WARPS_PER_BLOCK * warp_id;
    B_lane_ptr = (int4*) (B_warp_ptr + (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                 (lane_id % CHUNK_COPY_LINE_LANES);
    B_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
    for (size_t i = 0; i < B_smem_iters; ++i) {
        uint32_t B_smem_lane_addr =
            __cvta_generic_to_shared(&smem[B_smem_idx][0]) +
            ((lane_id % CHUNK_COPY_LINE_LANES +
              (B_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) /
                  SMEM_BANK_ROWS) %
             CHUNK_COPY_LINE_LANES) *
                THREAD_COPY_BYTES;

        CP_ASYNC_CG(B_smem_lane_addr, B_lane_ptr, THREAD_COPY_BYTES);

        B_lane_ptr =
            (int4*) ((float16_t*) B_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
        B_smem_idx += CHUNK_COPY_LINES_PER_WARP;
    }

    CP_ASYNC_COMMIT_GROUP();

    smem_store_idx = (smem_store_idx + 1) % K_STAGE;
    smem_store_off = smem_store_idx * smem_stage_off;

    A_smem_idx = smem_store_off + BLOCK_ROWS / WARPS_PER_BLOCK * warp_id;
    A_lane_ptr = (int4*) (A_warp_ptr + CHUNK_K * MMA_K +
                          (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                 (lane_id % CHUNK_COPY_LINE_LANES);
    A_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
    for (size_t i = 0; i < A_smem_iters; ++i) {
        uint32_t A_smem_lane_addr =
            __cvta_generic_to_shared(&smem[A_smem_idx][0]) +
            ((lane_id % CHUNK_COPY_LINE_LANES +
              (A_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) /
                  SMEM_BANK_ROWS) %
             CHUNK_COPY_LINE_LANES) *
                THREAD_COPY_BYTES;

        CP_ASYNC_CG(A_smem_lane_addr, A_lane_ptr, THREAD_COPY_BYTES);

        A_lane_ptr =
            (int4*) ((float16_t*) A_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
        A_smem_idx += CHUNK_COPY_LINES_PER_WARP;
    }

    B_smem_idx = smem_store_off + B_smem_idx_off +
                 BLOCK_COLS / WARPS_PER_BLOCK * warp_id;
    B_lane_ptr = (int4*) (B_warp_ptr + CHUNK_K * MMA_K +
                          (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                 (lane_id % CHUNK_COPY_LINE_LANES);
    B_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
    for (size_t i = 0; i < B_smem_iters; ++i) {
        uint32_t B_smem_lane_addr =
            __cvta_generic_to_shared(&smem[B_smem_idx][0]) +
            ((lane_id % CHUNK_COPY_LINE_LANES +
              (B_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) /
                  SMEM_BANK_ROWS) %
             CHUNK_COPY_LINE_LANES) *
                THREAD_COPY_BYTES;

        CP_ASYNC_CG(B_smem_lane_addr, B_lane_ptr, THREAD_COPY_BYTES);

        B_lane_ptr =
            (int4*) ((float16_t*) B_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
        B_smem_idx += CHUNK_COPY_LINES_PER_WARP;
    }

    CP_ASYNC_COMMIT_GROUP();

    smem_store_idx = (smem_store_idx + 1) % K_STAGE;
    smem_store_off = smem_store_idx * smem_stage_off;

    A_smem_idx = smem_store_off + BLOCK_ROWS / WARPS_PER_BLOCK * warp_id;
    A_lane_ptr = (int4*) (A_warp_ptr + 2 * CHUNK_K * MMA_K +
                          (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                 (lane_id % CHUNK_COPY_LINE_LANES);
    A_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
    for (size_t i = 0; i < A_smem_iters; ++i) {
        uint32_t A_smem_lane_addr =
            __cvta_generic_to_shared(&smem[A_smem_idx][0]) +
            ((lane_id % CHUNK_COPY_LINE_LANES +
              (A_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) /
                  SMEM_BANK_ROWS) %
             CHUNK_COPY_LINE_LANES) *
                THREAD_COPY_BYTES;

        CP_ASYNC_CG(A_smem_lane_addr, A_lane_ptr, THREAD_COPY_BYTES);

        A_lane_ptr =
            (int4*) ((float16_t*) A_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
        A_smem_idx += CHUNK_COPY_LINES_PER_WARP;
    }

    B_smem_idx = smem_store_off + B_smem_idx_off +
                 BLOCK_COLS / WARPS_PER_BLOCK * warp_id;
    B_lane_ptr = (int4*) (B_warp_ptr + 2 * CHUNK_K * MMA_K +
                          (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                 (lane_id % CHUNK_COPY_LINE_LANES);
    B_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
    for (size_t i = 0; i < B_smem_iters; ++i) {
        uint32_t B_smem_lane_addr =
            __cvta_generic_to_shared(&smem[B_smem_idx][0]) +
            ((lane_id % CHUNK_COPY_LINE_LANES +
              (B_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) /
                  SMEM_BANK_ROWS) %
             CHUNK_COPY_LINE_LANES) *
                THREAD_COPY_BYTES;

        CP_ASYNC_CG(B_smem_lane_addr, B_lane_ptr, THREAD_COPY_BYTES);

        B_lane_ptr =
            (int4*) ((float16_t*) B_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
        B_smem_idx += CHUNK_COPY_LINES_PER_WARP;
    }

    CP_ASYNC_COMMIT_GROUP();
    CP_ASYNC_WAIT_GROUP(2);

    __syncthreads();

    uint32_t RA[2][WARP_COL_TILES][4];
    uint32_t RB[2][WARP_ROW_TILES][2];

    size_t reg_store_idx = 0;
    size_t reg_load_idx = 1;

#pragma unroll
    for (size_t i = 0; i < WARP_COL_TILES; ++i) {
        size_t A_smem_idx = smem_load_off +
                            (warp_id / BLOCK_ROW_WARPS) * WARP_ROWS +
                            i * MMA_M;
        uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
            &smem[A_smem_idx + lane_id % 16]
                 [((lane_id / 16) * 8 +
                   (lane_id % 16 % (PERMUTED_COLS * SMEM_BANK_ROWS)) /
                       SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                  AB_SMEM_STRIDE]);

        LDMATRIX_X4(RA[reg_store_idx][i][0], RA[reg_store_idx][i][1],
                    RA[reg_store_idx][i][2], RA[reg_store_idx][i][3],
                    A_smem_lane_addr);
    }

#pragma unroll
    for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
        size_t B_smem_idx = smem_load_off + B_smem_idx_off +
                            (warp_id % BLOCK_ROW_WARPS) * WARP_COLS +
                            j * MMA_N;
        uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
            &smem[B_smem_idx + lane_id % 8]
                 [(((lane_id / 8) % 2) * 8 +
                   (lane_id % 8 % (PERMUTED_COLS * SMEM_BANK_ROWS)) /
                       SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                  AB_SMEM_STRIDE]);

        LDMATRIX_X2(RB[reg_store_idx][j][0], RB[reg_store_idx][j][1],
                    B_smem_lane_addr);
    }

#pragma unroll
    for (size_t tile_k = CHUNK_K * (K_STAGE - 1); tile_k < K_tiles;
         tile_k += CHUNK_K) {
        reg_store_idx ^= 1;
        reg_load_idx ^= 1;

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
            size_t A_smem_idx = smem_load_off +
                                (warp_id / BLOCK_ROW_WARPS) * WARP_ROWS +
                                i * MMA_M;
            uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
                &smem[A_smem_idx + lane_id % 16]
                     [(MMA_K + (lane_id / 16) * 8 +
                       (lane_id % 16 % (PERMUTED_COLS * SMEM_BANK_ROWS)) /
                           SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X4(RA[reg_store_idx][i][0], RA[reg_store_idx][i][1],
                        RA[reg_store_idx][i][2], RA[reg_store_idx][i][3],
                        A_smem_lane_addr);
        }

#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            size_t B_smem_idx = smem_load_off + B_smem_idx_off +
                                (warp_id % BLOCK_ROW_WARPS) * WARP_COLS +
                                j * MMA_N;
            uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
                &smem[B_smem_idx + lane_id % 8]
                     [(MMA_K + ((lane_id / 8) % 2) * 8 +
                       (lane_id % 8 % (PERMUTED_COLS * SMEM_BANK_ROWS)) /
                           SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X2(RB[reg_store_idx][j][0], RB[reg_store_idx][j][1],
                        B_smem_lane_addr);
        }

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
            for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
                // 实现波浪式（Zig-Zag）的矩阵计算顺序
                size_t j_s = (i % 2) ? (WARP_ROW_TILES - j - 1) : j;

                HMMA16816(RC[i][j_s][0], RC[i][j_s][1], RA[reg_load_idx][i][0],
                          RA[reg_load_idx][i][1], RA[reg_load_idx][i][2],
                          RA[reg_load_idx][i][3], RB[reg_load_idx][j_s][0],
                          RB[reg_load_idx][j_s][1], RC[i][j_s][0],
                          RC[i][j_s][1]);
            }
        }

        smem_store_idx = (smem_store_idx + 1) % K_STAGE;
        smem_store_off = smem_store_idx * smem_stage_off;

        A_smem_idx = smem_store_off + BLOCK_ROWS / WARPS_PER_BLOCK * warp_id;
        A_lane_ptr = (int4*) (A_warp_ptr + tile_k * MMA_K +
                              (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                     (lane_id % CHUNK_COPY_LINE_LANES);
        A_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
        for (size_t i = 0; i < A_smem_iters / CHUNK_K; ++i) {
            uint32_t A_smem_lane_addr =
                __cvta_generic_to_shared(&smem[A_smem_idx][0]) +
                ((lane_id % CHUNK_COPY_LINE_LANES +
                  (A_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) /
                      SMEM_BANK_ROWS) %
                 CHUNK_COPY_LINE_LANES) *
                    THREAD_COPY_BYTES;

            CP_ASYNC_CG(A_smem_lane_addr, A_lane_ptr, THREAD_COPY_BYTES);

            A_lane_ptr =
                (int4*) ((float16_t*) A_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
            A_smem_idx += CHUNK_COPY_LINES_PER_WARP;
        }

        B_smem_idx = smem_store_off + B_smem_idx_off +
                     BLOCK_COLS / WARPS_PER_BLOCK * warp_id;
        B_lane_ptr = (int4*) (B_warp_ptr + tile_k * MMA_K +
                              (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                     (lane_id % CHUNK_COPY_LINE_LANES);
        B_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
        for (size_t i = 0; i < B_smem_iters / CHUNK_K; ++i) {
            uint32_t B_smem_lane_addr =
                __cvta_generic_to_shared(&smem[B_smem_idx][0]) +
                ((lane_id % CHUNK_COPY_LINE_LANES +
                  (B_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) /
                      SMEM_BANK_ROWS) %
                 CHUNK_COPY_LINE_LANES) *
                    THREAD_COPY_BYTES;

            CP_ASYNC_CG(B_smem_lane_addr, B_lane_ptr, THREAD_COPY_BYTES);

            B_lane_ptr =
                (int4*) ((float16_t*) B_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
            B_smem_idx += CHUNK_COPY_LINES_PER_WARP;
        }

        smem_load_idx = (smem_load_idx + 1) % K_STAGE;
        smem_load_off = smem_load_idx * smem_stage_off;

#pragma unroll
        for (size_t i = (CHUNK_K - 1) * A_smem_iters / CHUNK_K;
             i < A_smem_iters; ++i) {
            uint32_t A_smem_lane_addr =
                __cvta_generic_to_shared(&smem[A_smem_idx][0]) +
                ((lane_id % CHUNK_COPY_LINE_LANES +
                  (A_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) /
                      SMEM_BANK_ROWS) %
                 CHUNK_COPY_LINE_LANES) *
                    THREAD_COPY_BYTES;

            CP_ASYNC_CG(A_smem_lane_addr, A_lane_ptr, THREAD_COPY_BYTES);

            A_lane_ptr =
                (int4*) ((float16_t*) A_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
            A_smem_idx += CHUNK_COPY_LINES_PER_WARP;
        }

#pragma unroll
        for (size_t i = (CHUNK_K - 1) * B_smem_iters / CHUNK_K;
             i < B_smem_iters; ++i) {
            uint32_t B_smem_lane_addr =
                __cvta_generic_to_shared(&smem[B_smem_idx][0]) +
                ((lane_id % CHUNK_COPY_LINE_LANES +
                  (B_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) /
                      SMEM_BANK_ROWS) %
                 CHUNK_COPY_LINE_LANES) *
                    THREAD_COPY_BYTES;

            CP_ASYNC_CG(B_smem_lane_addr, B_lane_ptr, THREAD_COPY_BYTES);

            B_lane_ptr =
                (int4*) ((float16_t*) B_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
            B_smem_idx += CHUNK_COPY_LINES_PER_WARP;
        }

        CP_ASYNC_COMMIT_GROUP();
        CP_ASYNC_WAIT_GROUP(2);

        __syncthreads();

        reg_store_idx ^= 1;
        reg_load_idx ^= 1;

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
            size_t A_smem_idx = smem_load_off +
                                (warp_id / BLOCK_ROW_WARPS) * WARP_ROWS +
                                i * MMA_M;
            uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
                &smem[A_smem_idx + lane_id % 16]
                     [((lane_id / 16) * 8 +
                       (lane_id % 16 % (PERMUTED_COLS * SMEM_BANK_ROWS)) /
                           SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X4(RA[reg_store_idx][i][0], RA[reg_store_idx][i][1],
                        RA[reg_store_idx][i][2], RA[reg_store_idx][i][3],
                        A_smem_lane_addr);
        }

#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            size_t B_smem_idx = smem_load_off + B_smem_idx_off +
                                (warp_id % BLOCK_ROW_WARPS) * WARP_COLS +
                                j * MMA_N;
            uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
                &smem[B_smem_idx + lane_id % 8]
                     [(((lane_id / 8) % 2) * 8 +
                       (lane_id % 8 % (PERMUTED_COLS * SMEM_BANK_ROWS)) /
                           SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X2(RB[reg_store_idx][j][0], RB[reg_store_idx][j][1],
                        B_smem_lane_addr);
        }

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
            for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
                size_t j_s = (i % 2) ? (WARP_ROW_TILES - j - 1) : j;

                HMMA16816(RC[i][j_s][0], RC[i][j_s][1], RA[reg_load_idx][i][0],
                          RA[reg_load_idx][i][1], RA[reg_load_idx][i][2],
                          RA[reg_load_idx][i][3], RB[reg_load_idx][j_s][0],
                          RB[reg_load_idx][j_s][1], RC[i][j_s][0],
                          RC[i][j_s][1]);
            }
        }
    }

#pragma unroll
    for (size_t k_step = 0; k_step < CHUNK_K; ++k_step) {
        reg_store_idx ^= 1;
        reg_load_idx ^= 1;

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
            size_t A_smem_idx = smem_load_off +
                                (warp_id / BLOCK_ROW_WARPS) * WARP_ROWS +
                                i * MMA_M;
            uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
                &smem[A_smem_idx + lane_id % 16]
                     [(((k_step + 1) % CHUNK_K) * MMA_K + (lane_id / 16) * 8 +
                       (lane_id % 16 % (PERMUTED_COLS * SMEM_BANK_ROWS)) /
                           SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X4(RA[reg_store_idx][i][0], RA[reg_store_idx][i][1],
                        RA[reg_store_idx][i][2], RA[reg_store_idx][i][3],
                        A_smem_lane_addr);
        }

#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            size_t B_smem_idx = smem_load_off + B_smem_idx_off +
                                (warp_id % BLOCK_ROW_WARPS) * WARP_COLS +
                                j * MMA_N;
            uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
                &smem[B_smem_idx + lane_id % 8]
                     [(((k_step + 1) % CHUNK_K) * MMA_K +
                       ((lane_id / 8) % 2) * 8 +
                       (lane_id % 8 % (PERMUTED_COLS * SMEM_BANK_ROWS)) /
                           SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X2(RB[reg_store_idx][j][0], RB[reg_store_idx][j][1],
                        B_smem_lane_addr);
        }

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
            for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
                size_t j_s = (i % 2) ? (WARP_ROW_TILES - j - 1) : j;

                HMMA16816(RC[i][j_s][0], RC[i][j_s][1], RA[reg_load_idx][i][0],
                          RA[reg_load_idx][i][1], RA[reg_load_idx][i][2],
                          RA[reg_load_idx][i][3], RB[reg_load_idx][j_s][0],
                          RB[reg_load_idx][j_s][1], RC[i][j_s][0],
                          RC[i][j_s][1]);
            }
        }

        if (k_step + 2 == CHUNK_K) {
            smem_load_idx = (smem_load_idx + 1) % K_STAGE;
            smem_load_off = smem_load_idx * smem_stage_off;

            CP_ASYNC_WAIT_GROUP(1);

            __syncthreads();
        }
    }

#pragma unroll
    for (size_t k_step = 0; k_step < CHUNK_K; ++k_step) {
        reg_store_idx ^= 1;
        reg_load_idx ^= 1;

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
            size_t A_smem_idx = smem_load_off +
                                (warp_id / BLOCK_ROW_WARPS) * WARP_ROWS +
                                i * MMA_M;
            uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
                &smem[A_smem_idx + lane_id % 16]
                     [(((k_step + 1) % CHUNK_K) * MMA_K + (lane_id / 16) * 8 +
                       (lane_id % 16 % (PERMUTED_COLS * SMEM_BANK_ROWS)) /
                           SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X4(RA[reg_store_idx][i][0], RA[reg_store_idx][i][1],
                        RA[reg_store_idx][i][2], RA[reg_store_idx][i][3],
                        A_smem_lane_addr);
        }

#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            size_t B_smem_idx = smem_load_off + B_smem_idx_off +
                                (warp_id % BLOCK_ROW_WARPS) * WARP_COLS +
                                j * MMA_N;
            uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
                &smem[B_smem_idx + lane_id % 8]
                     [(((k_step + 1) % CHUNK_K) * MMA_K +
                       ((lane_id / 8) % 2) * 8 +
                       (lane_id % 8 % (PERMUTED_COLS * SMEM_BANK_ROWS)) /
                           SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X2(RB[reg_store_idx][j][0], RB[reg_store_idx][j][1],
                        B_smem_lane_addr);
        }

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
            for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
                size_t j_s = (i % 2) ? (WARP_ROW_TILES - j - 1) : j;

                HMMA16816(RC[i][j_s][0], RC[i][j_s][1], RA[reg_load_idx][i][0],
                          RA[reg_load_idx][i][1], RA[reg_load_idx][i][2],
                          RA[reg_load_idx][i][3], RB[reg_load_idx][j_s][0],
                          RB[reg_load_idx][j_s][1], RC[i][j_s][0],
                          RC[i][j_s][1]);
            }
        }

        if (k_step + 2 == CHUNK_K) {
            smem_load_idx = (smem_load_idx + 1) % K_STAGE;
            smem_load_off = smem_load_idx * smem_stage_off;

            CP_ASYNC_WAIT_GROUP(0);

            __syncthreads();
        }
    }

#pragma unroll
    for (size_t k_step = 1; k_step < CHUNK_K; ++k_step) {
        reg_store_idx ^= 1;
        reg_load_idx ^= 1;

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
            size_t A_smem_idx = smem_load_off +
                                (warp_id / BLOCK_ROW_WARPS) * WARP_ROWS +
                                i * MMA_M;
            uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
                &smem[A_smem_idx + lane_id % 16]
                     [(k_step * MMA_K + (lane_id / 16) * 8 +
                       (lane_id % 16 % (PERMUTED_COLS * SMEM_BANK_ROWS)) /
                           SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X4(RA[reg_store_idx][i][0], RA[reg_store_idx][i][1],
                        RA[reg_store_idx][i][2], RA[reg_store_idx][i][3],
                        A_smem_lane_addr);
        }

#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            size_t B_smem_idx = smem_load_off + B_smem_idx_off +
                                (warp_id % BLOCK_ROW_WARPS) * WARP_COLS +
                                j * MMA_N;
            uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
                &smem[B_smem_idx + lane_id % 8]
                     [(k_step * MMA_K + ((lane_id / 8) % 2) * 8 +
                       (lane_id % 8 % (PERMUTED_COLS * SMEM_BANK_ROWS)) /
                           SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X2(RB[reg_store_idx][j][0], RB[reg_store_idx][j][1],
                        B_smem_lane_addr);
        }

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
            for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
                size_t j_s = (i % 2) ? (WARP_ROW_TILES - j - 1) : j;

                HMMA16816(RC[i][j_s][0], RC[i][j_s][1], RA[reg_load_idx][i][0],
                          RA[reg_load_idx][i][1], RA[reg_load_idx][i][2],
                          RA[reg_load_idx][i][3], RB[reg_load_idx][j_s][0],
                          RB[reg_load_idx][j_s][1], RC[i][j_s][0],
                          RC[i][j_s][1]);
            }
        }
    }

#pragma unroll
    for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            size_t j_s = (i % 2) ? (WARP_ROW_TILES - j - 1) : j;

            HMMA16816(RC[i][j_s][0], RC[i][j_s][1], RA[reg_store_idx][i][0],
                      RA[reg_store_idx][i][1], RA[reg_store_idx][i][2],
                      RA[reg_store_idx][i][3], RB[reg_store_idx][j_s][0],
                      RB[reg_store_idx][j_s][1], RC[i][j_s][0], RC[i][j_s][1]);
        }
    }

    __syncthreads();

#pragma unroll
    for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            float16_t* lane_ptr0 =
                smem_warp_tile_row_ptr +
                (i * MMA_M + lane_id / 4) * C_SMEM_STRIDE +
                ((warp_id % BLOCK_ROW_WARPS) * C_SMEM_OFFSET + j * MMA_N +
                 (lane_id % 4) * sizeof(uint32_t) / sizeof(float16_t) +
                 ((lane_id / 4) % 8) * PERMUTED_OFFSET) %
                    C_SMEM_STRIDE;
            float16_t* lane_ptr1 =
                smem_warp_tile_row_ptr +
                (i * MMA_M + lane_id / 4 + 8) * C_SMEM_STRIDE +
                ((warp_id % BLOCK_ROW_WARPS) * C_SMEM_OFFSET + j * MMA_N +
                 (lane_id % 4) * sizeof(uint32_t) / sizeof(float16_t) +
                 ((lane_id / 4 + 8) % 8) * PERMUTED_OFFSET) %
                    C_SMEM_STRIDE;

            *((uint32_t*) (lane_ptr0)) = RC[i][j][0];
            *((uint32_t*) (lane_ptr1)) = RC[i][j][1];
        }
    }

    __syncthreads();

#pragma unroll
    for (size_t i = 0; i < MMA_M; ++i) {
        *((int4*) (src_gmem_warp_stream_ptr + (i * 2 + lane_id / 16) * N) +
          lane_id % 16) =
            *((int4*) (smem_warp_stream_ptr +
                       (i * 2 + lane_id / 16) * C_SMEM_STRIDE) +
              (lane_id % 16 + (i * 2 + lane_id / 16) % 8) %
                  (C_SMEM_STRIDE * sizeof(float16_t) / THREAD_COPY_BYTES));
    }
}

PLAYGROUND_MATMUL_DEC(float16_t, 19, M, N, K, A, B, C)
{
    const int sharedMemSize =
        std::max((BLOCK_ROWS + BLOCK_COLS) * AB_SMEM_STRIDE *
                     sizeof(float16_t) * K_STAGE,
                 BLOCK_ROWS * C_SMEM_STRIDE * sizeof(float16_t));
    dim3 blockDim(32, 8);
    dim3 gridDim(BLOCK_STRIDE, div_ceil(M, BLOCK_ROWS),
                 div_ceil(N, BLOCK_COLS * BLOCK_STRIDE));
    const int MMA_M = 16, MMA_N = 8, MMA_K = 16;
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        hgemm_mma_stage_v2<MMA_M), MMA_N, MMA_K>,
        hipFuncAttributeMaxDynamicSharedMemorySize, 131072);
    hgemm_mma_stage_v2<MMA_M, MMA_N, MMA_K>
        <<<gridDim, blockDim, sharedMemSize>>>(A, B, C, M, N, K);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }
}

}