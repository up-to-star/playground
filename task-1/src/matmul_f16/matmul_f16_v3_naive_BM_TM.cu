#include "hip/hip_runtime.h"
#include "playground/matmul.hpp"
#include "playground/system.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define div_ceil(n, m) (((n) + (m) - 1) / (m))

#define READ_HALF2(pointer)                                                    \
    (*reinterpret_cast<const half2*>(std::addressof(pointer)))
#define WRITE_HALF2(pointer)                                                    \
    (*reinterpret_cast<half2*>(std::addressof(pointer)))

namespace playground
{

template <typename DType, const int BM = 128, const int BN = 128, const int BK = 8, const int TM = 8, const int TN = 8>
__global__ void hgemm_naive_v2(const DType* __restrict__ A,
                               const DType* __restrict__ B,
                               DType* __restrict__ C, const size_t M,
                               const size_t N, const size_t K)
{
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int tid = ty * blockDim.x + tx;
    __shared__ float16_t sa[BM][BK], sb[BK][BN];

    int load_a_smem_m = tid >> 1;  // tid/2, row of s_a
    int load_a_smem_k = (tid & 1) << 2;
    int load_b_smem_k = tid >> 5;
    int load_b_smem_n = (tid & 31) << 2;

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    if (load_a_gmem_m >= M || load_b_gmem_n >= N)
        return;

    float16_t r_c[TM][TN] = {{__float2half(0.0)}};

    for (size_t bk = 0; bk < div_ceil(K, BK); bk++) {
        int load_a_gmem_k = bk * BK + load_a_smem_k;
        int load_a_gmem_addr = load_a_gmem_m * K + load_a_gmem_k;
        WRITE_HALF2(sa[load_a_smem_m][load_a_smem_k + 0]) =
            READ_HALF2(A[load_a_gmem_addr + 0]);
        WRITE_HALF2(sa[load_a_smem_m][load_a_smem_k + 2]) =
            READ_HALF2(A[load_a_gmem_addr + 2]);

        int load_b_gmem_k = bk * BK + load_b_smem_k;
        int load_b_gmem_addr = load_b_gmem_k * N + load_b_gmem_n;

        WRITE_HALF2(sb[load_b_smem_k][load_b_smem_n + 0]) =
            READ_HALF2(B[load_b_gmem_addr + 0]);
        WRITE_HALF2(sb[load_b_smem_k][load_b_smem_n + 2]) =
            READ_HALF2(B[load_b_gmem_addr + 2]);

        __syncthreads();
#pragma unroll
        for (size_t k = 0; k < BK; k++) {
#pragma unroll
            for (size_t m = 0; m < TM; m++) {
#pragma unroll
                for (size_t n = 0; n < TN; n++) {
                    int comp_a_smem_m = ty * TM + m;
                    int comp_b_smem_n = tx * TN + n;
                    r_c[m][n] += sa[comp_a_smem_m][k] * sb[k][comp_b_smem_n];
                }
            }
        }
        __syncthreads();
    }

#pragma unroll
    for (size_t m = 0; m < TM; m++) {
        int store_c_gmem_m = by * BM + ty * TM + m;
#pragma unroll
        for (size_t n = 0; n < TN; n += 2) {
            int store_c_gmem_n = bx * BN + tx * TN + n;
            int store_c_gmem_addr = store_c_gmem_m * N + store_c_gmem_n;
            WRITE_HALF2(C[store_c_gmem_addr]) = READ_HALF2(r_c[m][n]);
        }
    }
}

PLAYGROUND_MATMUL_DEC(float16_t, 3, M, N, K, A, B, C)
{
    const int BM = 128, BN = 128;
    const int BK = 8;
    const int TM = 8, TN = 8;

    dim3 blockDim(BN / TN, BM / TM);
    dim3 gridDim(div_ceil(N, BN), div_ceil(M, BM));
    hgemm_naive_v2<float16_t, BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(A, B, C, M, N, K);
}
}