#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/library_types.h>

#include "playground/cublas_handle.hpp"
#include "playground/matmul.hpp"
#include "playground/system.hpp"

namespace playground
{
PLAYGROUND_MATMUL_DEC(float16_t, 1, m, n, k, A, B, C)
{
    const float16_t Alpha = 1.0F;
    const float16_t Beta = 0.0F;
    hipblasGemmEx(s_getCublasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                 &Alpha, B, HIP_R_16F, n, A, HIP_R_16F, k, &Beta, C,
                 HIP_R_16F, n, HIP_R_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}
}  // namespace playground
