#include "hip/hip_runtime.h"
#include "playground/system.hpp"
#include "playground/matmul.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <cstdio>
#include <hip/hip_runtime_api.h>

using namespace nvcuda;

#define WARP_SIZE 32
#define HOST_DEVICE_INLINE __device__ __host__ inline
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n)                                                 \
    asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
#define CP_ASYNC_CA(dst, src, bytes)                                           \
    asm volatile(                                                              \
        "cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst),     \
        "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes)                                           \
    asm volatile(                                                              \
        "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst),     \
        "l"(src), "n"(bytes))

HOST_DEVICE_INLINE
int div_ceil(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

namespace playground
{

template <typename DType, const int WMMA_M = 16, const int WMMA_N = 16,
          const int WMMA_K = 16, const int WMMA_TILE_M = 4,
          const int WMMA_TILE_N = 4, const int WARP_TILE_M = 4,
          const int WARP_TILE_N = 4, const int A_PAD = 0, const int B_PAD = 0,
          const int K_STAGE = 2, const bool BLOCK_SWIZZLE = false>
__global__ void __launch_bounds__(512)
    hgemm_wmma_mma4x4_warp4x4_dsmem_stage(const DType* __restrict__ A,
                                          const DType* __restrict__ B,
                                          DType* __restrict__ C, const int M,
                                          const int N, const int K)
{
    const int bx = blockIdx.x + ((int) BLOCK_SWIZZLE) * blockIdx.z * gridDim.x,
              by = blockIdx.y;
    const int NUM_K_TILES = div_ceil(K, WMMA_K);
    constexpr int BM = WMMA_M * WMMA_TILE_M * WARP_TILE_M; // 256
    constexpr int BN = WMMA_N * WMMA_TILE_N * WARP_TILE_N; // 256
    constexpr int BK = WMMA_K;
    if (bx >= N / BN || by >= M / BM) {
        return;
    }

    // 动态共享内存
    extern __shared__ DType smem[];

    DType* sa = smem;
    DType* sb = smem + K_STAGE * BM * (BK + A_PAD);
    constexpr int s_a_stage_offset = BM * (BK + A_PAD);
    constexpr int s_b_stage_offset = BK * (BN + B_PAD);

    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int warp_id = tid / WARP_SIZE;
    const int warp_m = warp_id >> 2;
    const int warp_n = warp_id & 3;

    // shared memeor 中的索引
    const int load_a_smem_m = tid >> 1;
    const int load_a_smem_k = (tid & 1) << 3;
    const int load_b_smem_k = tid >> 5;
    const int load_b_smem_n = (tid & 31) << 3;
    const int load_a_gmem_m = by * BM + load_a_smem_m;
    const int load_b_gmem_n = bx * BN + load_b_smem_n;

    if (load_a_gmem_m >= M || load_b_gmem_n >= N) {
        return;
    }

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, DType>
        c_frag[WARP_TILE_M][WARP_TILE_N];
#pragma unroll
    for (int i = 0; i < WARP_TILE_M; i++) {
#pragma unroll
        for (int j = 0; j < WARP_TILE_N; j++) {
            wmma::fill_fragment(c_frag[i][j], 0.0);
        }
    }

    uint32_t smem_a_base_ptr = __cvta_generic_to_shared(sa);
    uint32_t smem_b_base_ptr = __cvta_generic_to_shared(sb);

    // 预加载前几个数据
#pragma unroll
    for (int k = 0; k < K_STAGE - 1; k++) {
        const int load_a_gmem_k = k * WMMA_K + load_a_smem_k;
        const int load_a_gmem_addr = load_a_gmem_m * K + load_a_gmem_k;
        const int load_b_gmem_k = k * WMMA_K + load_b_smem_k;
        const int load_b_gmem_addr = load_b_gmem_k * N + load_b_gmem_n;

        uint32_t load_a_smem_ptr =
            smem_a_base_ptr + (k * s_a_stage_offset +
                               load_a_smem_m * (BK + A_PAD) + load_a_smem_k) *
                                  sizeof(DType);
        CP_ASYNC_CG(load_a_smem_ptr, &A[load_a_gmem_addr], 16);

        uint32_t load_b_smem_ptr =
            smem_b_base_ptr + (k * s_b_stage_offset +
                               load_b_smem_k * (BN + B_PAD) + load_b_smem_n) *
                                  sizeof(DType);
        CP_ASYNC_CG(load_b_smem_ptr, &B[load_b_gmem_addr], 16);

        CP_ASYNC_COMMIT_GROUP();
    }
    CP_ASYNC_WAIT_GROUP(K_STAGE - 2);
    __syncthreads();

#pragma unroll 32
    for (int k = K_STAGE - 1; k < NUM_K_TILES; k++) {
        const int smem_sel = (k + 1) % K_STAGE;
        const int smem_sel_next = k % K_STAGE;

        const int load_a_gmem_k = k * WMMA_K + load_a_smem_k;
        const int load_a_gmem_addr = load_a_gmem_m * K + load_a_gmem_k;
        const int load_b_gmem_k = k * WMMA_K + load_b_smem_k;
        const int load_b_gmem_addr = load_b_gmem_k * N + load_b_gmem_n;

        uint32_t load_a_smem_ptr =
            smem_a_base_ptr + (smem_sel_next * s_a_stage_offset +
                               load_a_smem_m * (BK + A_PAD) + load_a_smem_k) *
                                  sizeof(DType);
        CP_ASYNC_CG(load_a_smem_ptr, &A[load_a_gmem_addr], 16);
        uint32_t load_b_smem_ptr =
            smem_b_base_ptr + (smem_sel_next * s_b_stage_offset +
                               load_b_smem_k * (BN + B_PAD) + load_b_smem_n) *
                                  sizeof(DType);
        CP_ASYNC_CG(load_b_smem_ptr, &B[load_b_gmem_addr], 16);
        CP_ASYNC_COMMIT_GROUP();

        wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, DType,
                       wmma::row_major>
            a_frag[WARP_TILE_M];
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, DType,
                       wmma::row_major>
            b_frag[WARP_TILE_N];

#pragma unroll
        for (int i = 0; i < WARP_TILE_M; i++) {
            const int warp_a_smem_m =
                warp_m * WMMA_M * WARP_TILE_M + i * WMMA_M;
            DType* load_a_smem_frag_ptr = sa + smem_sel * s_a_stage_offset +
                                          warp_a_smem_m * (BK + A_PAD) + 0;
            wmma::load_matrix_sync(a_frag[i], load_a_smem_frag_ptr, BK + A_PAD);
        }

#pragma unroll
        for (int j = 0; j < WARP_TILE_N; j++) {
            const int warp_b_smem_n =
                warp_n * WMMA_N * WARP_TILE_N + j * WMMA_N;
            DType* load_b_smem_frag_ptr = sb + smem_sel * s_b_stage_offset +
                                          0 * (BN + B_PAD) + warp_b_smem_n;
            wmma::load_matrix_sync(b_frag[j], load_b_smem_frag_ptr, BN + B_PAD);
        }

#pragma unroll
        for (int i = 0; i < WARP_TILE_M; i++) {
#pragma unroll
            for (int j = 0; j < WARP_TILE_N; j++) {
                wmma::mma_sync(c_frag[i][j], a_frag[i], b_frag[j], c_frag[i][j]);
            }
        }

        CP_ASYNC_WAIT_GROUP(K_STAGE - 2);
        __syncthreads();
    }

    if (K_STAGE > 1) {
        CP_ASYNC_WAIT_GROUP(0);
        __syncthreads();
    }

    // 处理最后几个
    {
#pragma unroll
        for (int k = 0; k < K_STAGE - 1; k++) {
            const int stage_sel = ((NUM_K_TILES - (K_STAGE - 1) + k) % K_STAGE);
            wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, DType,
                           wmma::row_major>
                a_frag[WARP_TILE_M];
            wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, DType,
                           wmma::row_major>
                b_frag[WARP_TILE_N];

#pragma unroll
            for (int i = 0; i < WARP_TILE_M; i++) {
                const int warp_a_smem_m =
                    warp_m * WMMA_M * WARP_TILE_M + i * WMMA_M;
                DType* load_a_smem_frag_ptr = sa +
                                              stage_sel * s_a_stage_offset +
                                              warp_a_smem_m * (BK + A_PAD) + 0;
                wmma::load_matrix_sync(a_frag[i], load_a_smem_frag_ptr, BK + A_PAD);
            }

#pragma unroll
            for (int j = 0; j < WARP_TILE_N; j++) {
                const int warp_b_smem_n =
                    warp_n * WMMA_N * WARP_TILE_N + j * WMMA_N;
                DType* load_b_smem_frag_ptr = sb +
                                              stage_sel * s_b_stage_offset +
                                              0 * (BN + B_PAD) + warp_b_smem_n;
                wmma::load_matrix_sync(b_frag[j], load_b_smem_frag_ptr, BN + B_PAD);
            }

#pragma unroll
            for (int i = 0; i < WARP_TILE_M; i++) {
#pragma unroll
                for (int j = 0; j < WARP_TILE_N; j++) {
                    wmma::mma_sync(c_frag[i][j], a_frag[i], b_frag[j], c_frag[i][j]);
                }
            }
        }
    }

#pragma unroll
    for (int i = 0; i < WARP_TILE_M; i++) {
#pragma unroll
        for (int j = 0; j < WARP_TILE_N; j++) {
            const int store_c_gmem_m =
                by * BM + warp_m * WMMA_M * WARP_TILE_M + i * WMMA_M;
            const int store_c_gmem_n =
                bx * BN + warp_n * WMMA_N * WARP_TILE_N + j * WMMA_N;
            wmma::store_matrix_sync(C + store_c_gmem_m * N + store_c_gmem_n, c_frag[i][j], N, wmma::mem_row_major);
        }
    }
}


PLAYGROUND_MATMUL_SIG(float16_t, 12, M, N, K, A, B, C)
{
    constexpr int BM = 256, BN = 256;
    constexpr int BK = 16;
    constexpr int A_PAD = 8, B_PAD = 8;
    dim3 blockDim(32, 16);
    // dim3 gridDim(div_ceil(N, BN), div_ceil(M, BM));
    constexpr int K_STAGE = 3;
    size_t sharedMemSize =
        K_STAGE * (BM * (BK + A_PAD) + BK * (BN + B_PAD)) * sizeof(float16_t);
    constexpr int WMMA_M = 16, WMMA_N = 16, WMMA_K = 16;
    constexpr int WMMA_TILE_M = 4, WMMA_TILE_N = 4;
    constexpr int WARP_TILE_M = 4, WARP_TILE_N = 4;
    const int BX = div_ceil(N, BN), BY = div_ceil(M, BM);
    const int NSPLIT = 4096;
    int split_num = (N + NSPLIT - 1) / NSPLIT;
    // const int split_num = 1;
    dim3 gridDim((BX + split_num - 1) / split_num, BY, split_num);
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        hgemm_wmma_mma4x4_warp4x4_dsmem_stage<
            float16_t), WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N,
            WARP_TILE_M, WARP_TILE_N, A_PAD, B_PAD, K_STAGE, true>,
        hipFuncAttributeMaxDynamicSharedMemorySize, 98304);
    hgemm_wmma_mma4x4_warp4x4_dsmem_stage<float16_t, WMMA_M, WMMA_N, WMMA_K,
                                          WMMA_TILE_M, WMMA_TILE_N, WARP_TILE_M,
                                          WARP_TILE_N, A_PAD, B_PAD, K_STAGE, true>
        <<<gridDim, blockDim, sharedMemSize>>>(A, B, C, M, N, K);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
    }
}


}